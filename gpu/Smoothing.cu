
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define CPUonly

#ifdef CPUonly
void generateData(double vectY[], const double vectX[], size_t vectXSize)
{
  for (size_t i = 0; i < vectXSize; i++)
  {
    vectY[i] = sin(0.02 * vectX[i]) + sin(0.001 * vectX[i]) + 0.1 * (rand() / (1.0 * RAND_MAX));
  }
}
#else
__global__ void generateData(double vectY[], const double vectX[], size_t vectXSize)
{
  size_t i = threadIdx.x;
  if (i < vectXSize)
  {
    vectY[i] = sin(0.02 * vectX[i]) + sin(0.001 * vectX[i]) + 0.1 /** (rand() / (1.0 * RAND_MAX))*/;
  }
}
#endif

#ifdef CPUonly
void aproximateValues(double y[], double x[], double yest[], int smooth, size_t N)
{
  for (size_t i = 0; i < N; i++)
  {
#else
__global__ void aproximateValues(double y[], double x[], double yest[], int smooth, size_t N)
{
  size_t i = threadIdx.x;
  if (i < N)
  {
#endif
    double sumA = 0.0;
    double sumB = 0.0;
    double temp = 0.0;
    for (size_t j = 0; j < N; j++)
    {
      temp = exp((-1 * ((x[i] - x[j]) * (x[i] - x[j]))) / (2 * smooth * smooth));
      sumA = sumA + temp * y[j];
      sumB = sumB + temp;
    }
    yest[i] = sumA /sumB;
  }
}


int main()
{

  size_t N = 10000;  // numero de ponto no dataset
  int smooth = 4; // parametro de suavidade

  // criar o dataset de entrada
  double *x = (double*)malloc(N * sizeof(double));
  double *y = (double*)malloc(N * sizeof(double));
  double *yest = (double*)malloc(N * sizeof(double));

  for (size_t i = 0; i < N; i++) {
    x[i] = i / 10.0;
  }
  memset(yest, 0, N*sizeof(double));

#ifdef CPUonly
  generateData(y, x, N);
  aproximateValues(y, x, yest, smooth, N);
#else
  double* d_x;
  double* d_y;
  double* d_yest; //d_ quer dizer variável do dispositivo, convenção

  cudaMalloc(&d_x, N*sizeof(double));
  cudaMalloc(&d_y, N*sizeof(double));
  cudaMalloc(&d_yest, N*sizeof(double));

  cudaMemcpy(d_x, x, N*sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(d_yest, yest, N*sizeof(double), cudaMemcpyHostToDevice);

  generateData << <1, N >> > (d_y, d_x, N);
  aproximateValues << <1, N >> > (d_y, d_x, d_yest, smooth, N);

  cudaMemcpy(yest, d_yest, N*sizeof(double), cudaMemcpyDeviceToHost);
#endif

  for (size_t i = 0; i < N; i++)
  {
    printf("yest[%d] = %lf\n", i, yest[i]);
  }
}
