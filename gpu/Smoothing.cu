#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

//#define CPUonly
#define N 50000 // numero de ponto no dataset


void generateData(float vectY[], const float vectX[])
{
    for (size_t i = 0; i < N; i++)
        vectY[i] = sin(0.02 * vectX[i]) + sin(0.001 * vectX[i]) + 0.1 * (rand() / (1.0 * RAND_MAX));
}

float timeDiff(struct timespec tStart, struct timespec tEnd)
{
   struct timespec diff;

   diff.tv_sec  = tEnd.tv_sec  - tStart.tv_sec  - (tEnd.tv_nsec<tStart.tv_nsec?1:0);
   diff.tv_nsec = tEnd.tv_nsec - tStart.tv_nsec + (tEnd.tv_nsec<tStart.tv_nsec?1000000000:0);

   return ((float) diff.tv_sec) + ((float) diff.tv_nsec)/1e9;
}

void aproximateValuesCPU(float y[], float x[], float yest[], int smooth)
{
    for (size_t i = 0; i < N; i++)
    {
        float sumA = 0.0;
        float sumB = 0.0;
        float temp = 0.0;
        for (size_t j = 0; j < N; j++)
        {
            temp = exp((-1 * ((x[i] - x[j]) * (x[i] - x[j]))) / (2 * smooth * smooth));
            sumA = sumA + temp * y[j];
            sumB = sumB + temp;
        }
        yest[i] = sumA / sumB;
    }
}

__global__ void aproximateValuesGPU(float y[], float x[], float yest[], int smooth)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
    {
        float sumA = 0.0;
        float sumB = 0.0;
        float temp = 0.0;
        for (size_t j = 0; j < N; j++)
        {
            temp = exp((-1 * ((x[i] - x[j]) * (x[i] - x[j]))) / (2 * smooth * smooth));
            sumA = sumA + temp * y[j];
            sumB = sumB + temp;
        }
        yest[i] = sumA / sumB;
    }
}


int main()
{
    int smooth = 4; // parametro de suavidade

    struct timespec timeVect[7];
    float timeCPU, timeGPU[7];
    hipError_t err[] = { hipSuccess , hipSuccess , hipSuccess };

    // criar o dataset de entrada
    float *x = (float*)malloc(N * sizeof(float));
    float *y = (float*)malloc(N * sizeof(float));
    float *yest = (float*)malloc(N * sizeof(float));
    float *yestCPU = (float*)malloc(N * sizeof(float));

    // Verify that allocations succeeded
    if (x == NULL || y == NULL || yest == NULL || yestCPU == NULL )
    {
        fprintf(stderr, "Failed to allocate on the host!\n");
        exit(EXIT_FAILURE);
    }

    for (size_t i = 0; i < N; i++)
        x[i] = i / 10.0;

    generateData(y, x);

/* CPU */
    printf("Performing the computation on the CPU...\n");
    clock_gettime(CLOCK_REALTIME, &timeVect[0]);

    aproximateValuesCPU(y, x, yestCPU, smooth);

    clock_gettime(CLOCK_REALTIME, &timeVect[1]);
    timeCPU = timeDiff(timeVect[0],timeVect[1]);
    printf("cpu done ... execution took %.6f seconds\n", timeCPU);

    printf("\n----------------------------------------------------------------------------\n\n");
/* GPU */

    printf("Performing the computation on the GPU...\n");
    clock_gettime(CLOCK_REALTIME, &timeVect[0]);
    clock_gettime(CLOCK_REALTIME, &timeVect[1]);

    // initialize the device (just measure the time for the first call to the device)
    clock_gettime(CLOCK_REALTIME, &timeVect[0]);
    hipFree(0);
    clock_gettime(CLOCK_REALTIME, &timeVect[1]);

    printf(" ... Allocation of memory on the Device ...\n");
    float* d_x;
    float* d_y;
    float* d_yest; //d_ quer dizer variável do dispositivo, convenção

    err[0] = hipMalloc(&d_x, N*sizeof(float));
    err[1] = hipMalloc(&d_y, N*sizeof(float));
    err[2] = hipMalloc(&d_yest, N*sizeof(float));

    if ((err[0] != hipSuccess) || (err[1] != hipSuccess) || (err[2] != hipSuccess))
    {
        fprintf(stderr, "Failed to allocate on the Device\n");
        exit(EXIT_FAILURE);
    }

    clock_gettime(CLOCK_REALTIME, &timeVect[2]);

    printf(" ... Copying input data from the host memory to the CUDA device ...\n");
    err[0] = hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    err[1] = hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
    

    if ((err[0] != hipSuccess) || (err[1] != hipSuccess))
    {
        fprintf(stderr, "Failed to copy data to the device!\n");
        exit(EXIT_FAILURE);
    }

    clock_gettime(CLOCK_REALTIME, &timeVect[3]);

    printf(" ... CUDA kernel launch with %d blocks of %d threads ...\n", N/1024, 1024);
    // Máximo de 1024 threads concorrentes!

    int blockDim = 1024;
    int gridDim = N / 1024 + (N % 1024 > 0);
    aproximateValuesGPU <<< gridDim, blockDim>>>(d_y, d_x, d_yest, smooth);

    err[0] = hipGetLastError();
    if (err[0] != hipSuccess)
    {
        fprintf(stderr, "Failed to launch yest calculation kernel (error code %s)!\n", hipGetErrorString(err[0]));
        exit(EXIT_FAILURE);
    }

    clock_gettime(CLOCK_REALTIME, &timeVect[4]);

    printf("Copy output data from the CUDA device to the host memory\n");
    err[0] = hipMemcpy(yest, d_yest, N*sizeof(float), hipMemcpyDeviceToHost);
    if (err[0] != hipSuccess)
    {
        fprintf(stderr, "Failed to copy yest from device to host (error code %s)!\n", hipGetErrorString(err[0]));
        exit(EXIT_FAILURE);
    }
    clock_gettime(CLOCK_REALTIME, &timeVect[5]);

    err[0] = hipFree(d_x);
    err[1] = hipFree(d_y);
    err[2] = hipFree(d_yest);

    if ((err[0] != hipSuccess) || (err[1] != hipSuccess) || (err[2] != hipSuccess))
    {
        fprintf(stderr, "Failed to free device vectors!\n");
        exit(EXIT_FAILURE);
    }
    clock_gettime(CLOCK_REALTIME, &timeVect[6]);
    
    timeGPU[0] = timeDiff(timeVect[0],timeVect[1]);
    timeGPU[1] = timeDiff(timeVect[1],timeVect[2]);
    timeGPU[2] = timeDiff(timeVect[2],timeVect[3]);
    timeGPU[3] = timeDiff(timeVect[3],timeVect[4]);
    timeGPU[4] = timeDiff(timeVect[4],timeVect[5]);
    timeGPU[5] = timeDiff(timeVect[5],timeVect[6]);
    timeGPU[6] = timeDiff(timeVect[0],timeVect[6]);
    printf("gpu done  ... execution took %.6f seconds (speedup=%.3f), corresponging to:\n",timeGPU[6],timeCPU/timeGPU[6]);
    printf("          - first call to the device           -> %.6f seconds\n",timeGPU[0]);
    printf("          - allocation of memory on the device -> %.6f seconds\n",timeGPU[1]);
    printf("          - copying data from host to device   -> %.6f seconds\n",timeGPU[2]);
    printf("          - kernel execution on the device     -> %.6f seconds\n",timeGPU[3]);
    printf("          - copying data from device to host   -> %.6f seconds\n",timeGPU[4]);
    printf("          - freeing data on the device         -> %.6f seconds\n",timeGPU[5]);
    printf("----------------------------------------------------------------------------\n");

    FILE *inputFile = fopen("input", "w");
    FILE *cpuFile = fopen("cpu-yest", "w");
    FILE *gpuFile = fopen("gpu-yest", "w");

    if(cpuFile == NULL || gpuFile == NULL) {
        printf("ficheiro não pode ser criado\n");
    }

    for (size_t i = 0; i < N; i++) {
        fprintf(inputFile, "%.30f\n", y[i]);
        fprintf(cpuFile, "%.30f\n", yestCPU[i]);
        fprintf(gpuFile, "%.30f\n", yest[i]);
    } 

    int errors = 0;
    float avgerror=0;
    for (size_t i = 0; i < N; i++)
    {
        if (yest[i] != yestCPU[i])
        {
            errors++;
            avgerror +=  abs(yestCPU[i]-yest[i]);
        }
    }    

    if(errors)
        printf("\nTest Failed:\n\t %d errors found! (out of %d points, %d%%)\n\t average error: %.20f\n", errors, N, errors*100/N, avgerror/errors);    
    else
        printf("Test Passes\n");
    
    free(x);
    free(y);
    free(yest);
    free(yestCPU);  

    // Reset the device and exit
    err[0] = hipDeviceReset();

    if (err[0] != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err[0]));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
