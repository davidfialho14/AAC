#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//#define CPUonly

void generateData(double vectY[], const double vectX[], size_t vectXSize)
{
    for (size_t i = 0; i < vectXSize; i++)
        vectY[i] = sin(0.02 * vectX[i]) + sin(0.001 * vectX[i]) + 0.1 * (rand() / (1.0 * RAND_MAX));
}

void aproximateValuesCPU(double y[], double x[], double yest[], int smooth, size_t N)
{
    for (size_t i = 0; i < N; i++)
    {
        double sumA = 0.0;
        double sumB = 0.0;
        double temp = 0.0;
        for (size_t j = 0; j < N; j++)
        {
            temp = exp((-1 * ((x[i] - x[j]) * (x[i] - x[j]))) / (2 * smooth * smooth));
            sumA = sumA + temp * y[j];
            sumB = sumB + temp;
        }
        yest[i] = sumA / sumB;
    }
}

__global__ void aproximateValuesGPU(double y[], double x[], double yest[], int smooth, size_t N)
{
    size_t i = threadIdx.x;
    if (i < N)
    {
        double sumA = 0.0;
        double sumB = 0.0;
        double temp = 0.0;
        for (size_t j = 0; j < N; j++)
        {
            temp = exp((-1 * ((x[i] - x[j]) * (x[i] - x[j]))) / (2 * smooth * smooth));
            sumA = sumA + temp * y[j];
            sumB = sumB + temp;
        }
        yest[i] = sumA / sumB;
    }
}


int main()
{

    size_t N = 10000;  // numero de ponto no dataset
    int smooth = 4; // parametro de suavidade

    // criar o dataset de entrada
    double *x = (double*)malloc(N * sizeof(double));
    double *y = (double*)malloc(N * sizeof(double));
    double *yest = (double*)malloc(N * sizeof(double));

    for (size_t i = 0; i < N; i++)
        x[i] = i / 10.0;

    generateData(y, x, N);

#ifdef CPUonly  
    aproximateValuesCPU(y, x, yest, smooth, N);

#else
    double* d_x;
    double* d_y;
    double* d_yest; //d_ quer dizer variável do dispositivo, convenção

    hipMalloc(&d_x, N*sizeof(double));
    hipMalloc(&d_y, N*sizeof(double));
    hipMalloc(&d_yest, N*sizeof(double));

    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);

    // Máximo de 1024 threads concorrentes!
    for (size_t k = 0; k < N; k += 1024)
        aproximateValuesGPU << <1, (N - k > 1024 ? 1024 : N - k) >> >(d_y + k, d_x + k, d_yest + k, smooth, N);

    hipMemcpy(yest, d_yest, N*sizeof(double), hipMemcpyDeviceToHost);
    printf("gpu done\n");
#endif

    double *yest2 = (double*)malloc(N * sizeof(double));
    aproximateValuesCPU(y, x, yest2, smooth, N);
    printf("cpu done\n");

#ifndef CPUonly
    for (size_t i = 0; i < N; i++)
    {
        if (yest[i] != yest2[i])
            printf(":( > yest2[%d] = %lf\n", i, yest2[i]);
        printf("yest[%d] = %lf\n", i, yest[i]);
    }
#endif

    free(x);
    free(y);
    free(yest);
    free(yest2);

#ifndef CPUonly
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_yest);
#endif

}